#include "ILayer.h"

size_t ILayer::get_neuron_count()
{
	return neuron_count;
}

void ILayer::set_neuron_count(size_t neuron_count)
{
	this->neuron_count = neuron_count;
	connections->neuron_count = neuron_count;
}

void ILayer::initialize_fields(size_t connection_count, size_t neuron_count, bool initialize_connection_associated_gradient_count)
{
	size_t* neuron_gradients_starts = new size_t[neuron_count];
	size_t* connection_associated_gradient_counts = 0;
	if (initialize_connection_associated_gradient_count)
		connection_associated_gradient_counts = new size_t[neuron_count];
	size_t gradient_count = 0;
	for (size_t i = 0; i < neuron_count; i++)
	{
		size_t neuron_connection_count = connections->get_connection_count_at(i);

		if (initialize_connection_associated_gradient_count)
			connection_associated_gradient_counts[i] = neuron_connection_count + 1;
		neuron_gradients_starts[i] = gradient_count;
		
		gradient_count += neuron_connection_count + 1 + gradients_per_neuron;
	}

	hipMalloc(&this->neuron_gradients_starts, sizeof(size_t) * neuron_count);
	hipMemcpy(this->neuron_gradients_starts, neuron_gradients_starts, sizeof(size_t) * neuron_count, hipMemcpyHostToDevice);
	delete[] neuron_gradients_starts;

	if (initialize_connection_associated_gradient_count)
	{
		hipMalloc(&this->connection_associated_gradient_counts, sizeof(size_t) * neuron_count);
		hipMemcpy(this->connection_associated_gradient_counts, connection_associated_gradient_counts, sizeof(size_t) * neuron_count, hipMemcpyHostToDevice);
		delete[] connection_associated_gradient_counts;
	}

	layer_specific_initialize_fields(connection_count, neuron_count);
	hipDeviceSynchronize();
}

void ILayer::layer_specific_initialize_fields(size_t connection_count, size_t neuron_count)
{
}

void ILayer::ILayerClone(ILayer* base_layer)
{
	IConnections* cloned_connections = connections->connections_specific_clone();
	connections->IConnections_clone(cloned_connections);
	base_layer->connections = cloned_connections;

	base_layer->set_neuron_count(get_neuron_count());

	base_layer->execution_values_per_neuron = execution_values_per_neuron;
	
	base_layer->layer_derivative_count = layer_derivative_count;
	base_layer->derivatives_per_neuron = derivatives_per_neuron;

	base_layer->layer_gradient_count = layer_gradient_count;

	base_layer->hidden_states_per_neuron = hidden_states_per_neuron;

	base_layer->optimizer = host_clone_optimizer(optimizer);
	
	hipMalloc(&base_layer->neuron_gradients_starts, sizeof(size_t) * get_neuron_count());
	if (connection_associated_gradient_counts)
		hipMalloc(&base_layer->connection_associated_gradient_counts, sizeof(size_t) * get_neuron_count());
	hipDeviceSynchronize();

	hipMemcpy(base_layer->neuron_gradients_starts, neuron_gradients_starts, sizeof(size_t) * get_neuron_count(), hipMemcpyDeviceToDevice);
	if (connection_associated_gradient_counts)
		hipMemcpy(base_layer->connection_associated_gradient_counts, connection_associated_gradient_counts, sizeof(size_t) * neuron_count, hipMemcpyDeviceToDevice);
}

void ILayer::save(FILE* file)
{
	fwrite(&neuron_count, sizeof(size_t), 1, file);
	fwrite(&execution_values_per_neuron, sizeof(size_t), 1, file);
	fwrite(&layer_derivative_count, sizeof(size_t), 1, file);
	fwrite(&derivatives_per_neuron, sizeof(size_t), 1, file);
	fwrite(&layer_gradient_count, sizeof(size_t), 1, file);

	bool contains_connection_gradient_counts = connection_associated_gradient_counts != 0;
	fwrite(&contains_connection_gradient_counts, sizeof(bool), 1, file);
	
	save_array(neuron_gradients_starts, neuron_count, file, true);

	if (contains_connection_gradient_counts)
		save_array(connection_associated_gradient_counts, neuron_count, file, true);

	host_save_optimizer(file, optimizer);

	specific_save(file);
}

void ILayer::ILayer_load(FILE* file)
{
	fread(&neuron_count, sizeof(size_t), 1, file);
	fread(&execution_values_per_neuron, sizeof(size_t), 1, file);
	fread(&layer_derivative_count, sizeof(size_t), 1, file);
	fread(&derivatives_per_neuron, sizeof(size_t), 1, file);
	fread(&layer_gradient_count, sizeof(size_t), 1, file);

	bool contains_connection_associated_gradient_counts = 0;
	fread(&contains_connection_associated_gradient_counts, sizeof(bool), 1, file);

	load_array<size_t>(neuron_count, file, true);

	if (contains_connection_associated_gradient_counts)
		load_array<size_t>(neuron_count, file, true);

	optimizer = host_load_optimizer(file);
}

void ILayer::deallocate()
{
	call_Optimizer_destructor kernel(1, 1) (optimizer);
	hipDeviceSynchronize();

	connections->deallocate();
	layer_specific_deallocate();
	hipDeviceSynchronize();
	delete connections;
}

void ILayer::layer_specific_deallocate()
{

}

void ILayer::mutate_fields(evolution_metadata evolution_values)
{
}

void ILayer::add_neuron(size_t previous_layer_length, size_t previous_layer_activations_start, float previous_layer_connection_probability, size_t min_connections)
{
	size_t added_connection_count = connections->connection_count;
	connections->add_neuron(previous_layer_length, previous_layer_activations_start, previous_layer_connection_probability, min_connections);
	added_connection_count = connections->connection_count - added_connection_count;

	if (connection_associated_gradient_counts)
		connection_associated_gradient_counts = cuda_push_back(connection_associated_gradient_counts, sizeof(size_t) * neuron_count, 1 + added_connection_count, true);

	if (neuron_gradients_starts)
	{
		size_t* tmp_neuron_gradients_starts = new size_t[neuron_count + 1];
		hipMemcpy(tmp_neuron_gradients_starts, neuron_gradients_starts, sizeof(size_t) * neuron_count, hipMemcpyDeviceToHost);
		tmp_neuron_gradients_starts[neuron_count] = tmp_neuron_gradients_starts[neuron_count - 1] + 1 + connections->get_connection_count_at(neuron_count - 1) + gradients_per_neuron;

		hipFree(neuron_gradients_starts);
		hipMalloc(&neuron_gradients_starts, sizeof(size_t) * (neuron_count + 1));
		hipMemcpy(neuron_gradients_starts, tmp_neuron_gradients_starts, sizeof(size_t) * (neuron_count + 1), hipMemcpyHostToDevice);
		delete[] tmp_neuron_gradients_starts;
	}

	layer_derivative_count += derivatives_per_neuron;
	layer_gradient_count += added_connection_count + gradients_per_neuron + 1;

	layer_specific_add_neuron();

	set_neuron_count(neuron_count + 1);
}

void ILayer::layer_specific_add_neuron()
{

}

void ILayer::adjust_to_added_neuron(size_t added_neuron_i, float connection_probability)
{
	auto added_connections_neuron_i = std::vector<size_t>();
	connections->adjust_to_added_neuron(added_neuron_i, connection_probability, &added_connections_neuron_i);
	for (size_t i = 0; i < added_connections_neuron_i.size(); i++)
	{
		layer_gradient_count++;
		size_t added_connection_neuron_i = added_connections_neuron_i[i];
		size_t remaining_neuron_count = neuron_count - added_connection_neuron_i - 1;
		if (remaining_neuron_count)
		{
			if (connection_associated_gradient_counts)
				add_to_array kernel(1, 1) (
					connection_associated_gradient_counts + added_connection_neuron_i, 1, 1
				);
			if (neuron_gradients_starts)
				add_to_array kernel(remaining_neuron_count / 32 + (remaining_neuron_count % 32 > 0), 32) (
					neuron_gradients_starts + added_connection_neuron_i + 1, remaining_neuron_count, 1
				);
		}
	}
}

void ILayer::remove_neuron(size_t layer_neuron_i)
{
	size_t removed_connection_count = connections->connection_count;
	connections->remove_neuron(layer_neuron_i);
	removed_connection_count -= connections->connection_count;

	size_t removed_gradients = removed_connection_count + gradients_per_neuron + 1;
	layer_gradient_count -= removed_gradients;
	layer_derivative_count -= derivatives_per_neuron;

	if (neuron_gradients_starts)
	{
		neuron_gradients_starts = 
			cuda_remove_elements(neuron_gradients_starts, neuron_count, layer_neuron_i, 1, true);
		
		size_t after_deletion_neuron_count = get_neuron_count() - layer_neuron_i - 1;
		if (after_deletion_neuron_count)
			add_to_array kernel(after_deletion_neuron_count / 32 + (after_deletion_neuron_count % 32 > 0), 32) (
				neuron_gradients_starts + layer_neuron_i, after_deletion_neuron_count, -(int)removed_gradients
			);
	}

	if (connection_associated_gradient_counts)
		connection_associated_gradient_counts =
			cuda_remove_elements(connection_associated_gradient_counts, neuron_count, layer_neuron_i, 1, true);
	hipDeviceSynchronize();

	layer_specific_remove_neuron(layer_neuron_i);

	set_neuron_count(neuron_count - 1);
}

void ILayer::layer_specific_remove_neuron(size_t layer_neuron_i)
{
}

void ILayer::adjust_to_removed_neuron(size_t neuron_i)
{
	auto removed_connections_neuron_i = std::vector<size_t>();
	connections->adjust_to_removed_neuron(neuron_i, &removed_connections_neuron_i);
	for (size_t i = 0; i < removed_connections_neuron_i.size(); i++)
	{
		layer_gradient_count--;
		size_t removed_connection_neuron_i = removed_connections_neuron_i[i];
		size_t remaining_neuron_count = neuron_count - removed_connection_neuron_i - 1;
		if (remaining_neuron_count)
		{
			if (connection_associated_gradient_counts)
				add_to_array kernel(1, 1) (
					connection_associated_gradient_counts + removed_connection_neuron_i, 1, -1
				);
			if (neuron_gradients_starts)
				add_to_array kernel(remaining_neuron_count / 32 + (remaining_neuron_count % 32 > 0), 32) (
					neuron_gradients_starts + removed_connection_neuron_i + 1, remaining_neuron_count, -1
				);
		}
	}
}

void ILayer::delete_memory()
{
}
