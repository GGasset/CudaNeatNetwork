#include "hip/hip_runtime.h"

#include "Optimizer_init.h"

__device__ IOptimizer* initialize_optimizer(optimizers_enum optimizer, size_t parameter_count)
{
	IOptimizer* out = 0;

	switch (optimizer)
	{
	case no_optimizer:
		out = new IOptimizer();
		break;
	case Adam:
		out = new AdamOptimizer();
		break;
	default:
		return (0);
	}
	out->alloc_optimizer_values(parameter_count, false);
	return (out);
}

__global__ void global_optimizer_init(optimizers_enum optimizer, IOptimizer** out, size_t parameter_count)
{
	*out = initialize_optimizer(optimizer, parameter_count);
}

__host__ IOptimizer* host_optimizer_init(optimizers_enum optimizer, size_t parameter_count)
{
	IOptimizer** tmp = 0;
	hipMalloc(&tmp, sizeof(IOptimizer*));
	global_optimizer_init kernel(1, 1) (optimizer, tmp, parameter_count);
	hipDeviceSynchronize();

	IOptimizer* out = 0;
	hipMemcpy(&out, tmp, sizeof(IOptimizer*), hipMemcpyDeviceToHost);
	hipFree(tmp);
	return (out);
}

__global__ void call_Optimizer_destructor(IOptimizer *optimizer)
{
	if (!optimizer)
		return ;
	optimizer->cleanup();
}

__global__ void get_optimizer_data_buffer(IOptimizer* optimizer, void** out_buffer, size_t *buff_len)
{
	if (!out_buffer || !buff_len) return;

	size_t values_per_paramater = optimizer->values_per_parameter;
	size_t param_count = optimizer->parameter_count;
	size_t value_count = values_per_paramater * param_count;

	size_t header_size = sizeof(optimizers_enum) + sizeof(size_t) * 2;
	size_t buff_size = header_size + sizeof(field_t) * value_count;
	char* out = 0;
	hipMalloc(&out, buff_size);
	if (!out) return;

	memcpy(out, optimizer, header_size);
	if (optimizer->optimizer_values && value_count)
		memcpy(out + header_size, optimizer->optimizer_values, sizeof(field_t) * value_count);

	*out_buffer = out;
	*buff_len = buff_size;
}

__host__ void host_save_optimizer(FILE* file, IOptimizer* optimizer)
{
	void** device_buff = 0;
	size_t* device_buff_len = 0;

	hipMalloc(&device_buff, sizeof(char *));
	hipMalloc(&device_buff_len, sizeof(size_t));
	
	get_optimizer_data_buffer kernel(1, 1) (optimizer, device_buff, device_buff_len);
	hipDeviceSynchronize();

	size_t buff_len = 0;
	hipMemcpy(&buff_len, device_buff_len, sizeof(size_t), hipMemcpyDeviceToHost);
	hipFree(device_buff_len);

	void* buff = new char[buff_len];
	hipMemcpy(buff, device_buff, buff_len, hipMemcpyDeviceToHost);
	fwrite(buff, 1, buff_len, file);
	delete[] buff;
}
