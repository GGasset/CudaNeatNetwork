#include "hip/hip_runtime.h"

#include "IOptimizer.h"

__device__ void IOptimizer::alloc_optimizer_values(size_t param_count, bool copy_old_values)
{
	size_t old_param_count = parameter_count;
	field_t* old_optimizer_values = optimizer_values;
	parameter_count = param_count;

	hipMalloc(&optimizer_values, sizeof(field_t) * param_count * values_per_parameter);
	initialize_optimizer_values(optimizer_values);
	if (copy_old_values && old_optimizer_values && optimizer_values)
		hipMemcpy(optimizer_values, old_optimizer_values, sizeof(field_t) * h_min(param_count, old_param_count) * values_per_parameter, hipMemcpyDeviceToDevice);
	hipFree(old_optimizer_values);
}

__device__ void IOptimizer::initialize_optimizer_values(field_t* values)
{
	hipMemset(values, 0, sizeof(field_t) * parameter_count * values_per_parameter);
}

__device__ void IOptimizer::cleanup()
{
	if (optimizer_values)
		hipFree(optimizer_values);
	optimizer_values = 0;
	parameter_count = 0;
}

__device__ void IOptimizer::hyperparameter_subtract_gradient(field_t* parameter, data_t gradient, size_t layer_parameter_i, gradient_hyperparameters hyperparameters)
{
	gradient = device_closest_to_zero(gradient, abs(hyperparameters.gradient_clip) * (1 - 2 * (gradient <= 0)));
	gradient *= hyperparameters.learning_rate;
	subtract_gradient(parameter, gradient, layer_parameter_i, hyperparameters);
}

__device__ void IOptimizer::subtract_gradient(field_t* parameter, data_t gradient, size_t layer_parameter_i, gradient_hyperparameters hyperparameters)
{
	atomicAdd(parameter, -gradient);
}
